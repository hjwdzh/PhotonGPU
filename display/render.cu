#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// Utilities and system includes

#include <hip/hip_runtime_api.h>
#include "../scene/world.h"

#define PATH_DEPTH 5
#define PATH_MAX_DEPTH 3
#define NUM_SAMPLE 1

__device__ __host__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}

__device__ __host__
float rayIntersectsTriangle(glm::vec3 p, glm::vec3 d,
glm::vec3 v0, glm::vec3 v1, glm::vec3 v2, float& u, float& v) {

	glm::vec3 e1 = v1 - v0;
	glm::vec3 e2 = v2 - v0;
	glm::vec3 h = glm::cross(d, e2);
	float a = glm::dot(e1, h);

	if (a > -0.00001 && a < 0.00001)
		return -1;

	float f = 1 / a;
	glm::vec3 s = p - v0;
	u = f * glm::dot(s, h);

	if (u < -1e-5 || u > 1+1e-5)
		return -1;

	glm::vec3 q = glm::cross(s, e1);
	v = f * glm::dot(d, q);

	if (v < -1e-5 || u + v > 1 + 1e-5)
		return -1;

	// at this stage we can compute t to find out where
	// the intersection point is on the line
	float t = f * glm::dot(e2, q);

	if (t > 0.00001) // ray intersection
		return t;

	else // this means that there is a line intersection
		// but not a ray intersection
		return -1;
}

__device__ __host__
float tracing(glm::vec3 ray_o_o, glm::vec3 ray_t_o, float shadow, int& tri, int& obj, glm::vec4& hit_point, glm::vec2& uv, glm::vec4& normal,
	InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object) {
	float depth = 1e30;
	obj = -1;
	tri = -1;
	int j = 0;
	for (int k = 0; k < num_object; ++k) {

		glm::vec3& x = instanceData[k].offset;
		glm::vec3& axisX = instanceData[k].axisX;
		glm::vec3& axisY = instanceData[k].axisY;
		glm::vec3& axisZ = glm::cross(axisX, axisY);
		glm::vec3& scale = instanceData[k].scale;

		glm::mat4 rotate = glm::mat4(glm::vec4(axisX, 0), glm::vec4(axisY, 0), glm::vec4(axisZ, 0), glm::vec4(0, 0, 0, 1));
		glm::mat4 convert = glm::mat4(glm::vec4(1, 0, 0, 0), glm::vec4(0, 1, 0, 0), glm::vec4(0, 0, 1, 0), glm::vec4(x, 1))
			* rotate
			* glm::mat4(glm::vec4(scale.x, 0, 0, 0), glm::vec4(0, scale.y, 0, 0), glm::vec4(0, 0, scale.z, 0), glm::vec4(0, 0, 0, 1));
		glm::mat4 inv_convert = glm::mat4(glm::vec4(1 / scale.x, 0, 0, 0), glm::vec4(0, 1 / scale.y, 0, 0), glm::vec4(0, 0, 1 / scale.z, 0), glm::vec4(0, 0, 0, 1))
			* glm::transpose(rotate)
			* glm::mat4(glm::vec4(1, 0, 0, 0), glm::vec4(0, 1, 0, 0), glm::vec4(0, 0, 1, 0), glm::vec4(-x, 1));

		glm::vec4 ray_o4 = (inv_convert * glm::vec4(ray_o_o, 1));
		glm::vec4 ray_t4 = inv_convert * glm::vec4(ray_t_o, 0);
		float len = glm::length(ray_t4);
		glm::vec3 ray_o(ray_o4.x, ray_o4.y, ray_o4.z);
		glm::vec3 ray_t(ray_t4.x / len, ray_t4.y / len, ray_t4.z / len);
		int next_object = instanceData[k].s;
		while (j < next_object) {
			glm::vec3& v1 = vertexBuffer[j];
			glm::vec3& v2 = vertexBuffer[j + 1];
			glm::vec3& v3 = vertexBuffer[j + 2];
			float u, v;
			float t = rayIntersectsTriangle(ray_o, ray_t, v1, v2, v3, u, v) / len;
			if (t > 0 && t < depth) {
				depth = t;
				hit_point = (convert * (ray_o4 + ray_t4 * depth));
				if (shadow >= 0) {
					if (t < shadow) {
						return t;
					}
				}
				else {
					obj = k;
					tri = j;
					glm::vec3& n1 = normalBuffer[j];
					glm::vec3& n2 = normalBuffer[j + 1];
					glm::vec3& n3 = normalBuffer[j + 2];
					normal = (convert * glm::vec4(u * (n2 - n1) + v * (n3 - n1) + n1, 0));
					glm::vec2& uv1 = texBuffer[j];
					glm::vec2& uv2 = texBuffer[j + 1];
					glm::vec2& uv3 = texBuffer[j + 2];
					uv = uv1 + u * (uv2 - uv1) + v * (uv3 - uv1);
				}
			}
			j += 3;
		}
	}
	normal = normalize(normal);
	return depth;
}

__device__ __host__
glm::vec3 fetchTex(glm::vec2 uv, int objIndex, uchar3* imagesBuffer, glm::ivec3* imageOffsetBuffer)
{
	glm::ivec3& info = imageOffsetBuffer[objIndex];
	int offset = info.x;
	int iy = info.y;
	int ix = info.z;
	float x = ix * uv.x;
	float y = iy * uv.y;
	int lx = x, ly = y;
	int rx = lx + 1, ry = ly + 1;
	float wx = x - lx, wy = y - ly;
	if (lx < 0)
		lx += wx;
	if (ly < 0)
		ly += wy;
	if (rx >= ix)
		rx -= ix;
	if (ry >= iy)
		ry -= iy;
	int ind1 = offset + ly * ix + lx;
	int ind2 = offset + ly * ix + rx;
	int ind3 = offset + ry * ix + lx;
	int ind4 = offset + ry * ix + rx;
	uchar3& c1 = imagesBuffer[ind1];
	uchar3& c2 = imagesBuffer[ind2];
	uchar3& c3 = imagesBuffer[ind3];
	uchar3& c4 = imagesBuffer[ind4];
	float cx = (c1.x * (1 - wx) + c2.x * wx) * (1 - wy) + (c3.x * (1 - wx) + c4.x * wx) * wy;
	float cy = (c1.y * (1 - wx) + c2.y * wx) * (1 - wy) + (c3.y * (1 - wx) + c4.y * wx) * wy;
	float cz = (c1.z * (1 - wx) + c2.z * wx) * (1 - wy) + (c3.z * (1 - wx) + c4.z * wx) * wy;
	return glm::vec3(cz, cy, cx);
}

__device__ __host__
glm::vec3 lighting(glm::vec3 start_camera, glm::vec3 point, glm::vec3 normal, int tri_index, glm::vec2 uv, int obj_index,
	InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object,
	int num_direct_light, glm::vec3* direct_lights, glm::vec3* direct_lights_color,
	int num_point_light, glm::vec3* point_lights, glm::vec3* point_lights_color, glm::vec3 ambient,
	uchar3* imagesBuffer, glm::ivec3* imageOffsetBuffer, glm::vec3& orig_color) {
	float kd = instanceData[obj_index].kd;
	float ks = instanceData[obj_index].ks;//texture2D(materialSampler, vec2(1.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
	float ka = instanceData[obj_index].ka;// texture2D(materialSampler, vec2(16.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
	float alpha = instanceData[obj_index].alpha;// texture2D(materialSampler, vec2(20.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
	
	orig_color = fetchTex(uv, obj_index, imagesBuffer, imageOffsetBuffer);
//	int tex = int(0.1 + texture2D(materialSampler, vec2(2.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r);
//	orig_color = texture2D(renderSampler[tex], uv).rgb;
	glm::vec3 color = ka * orig_color * ambient;
	glm::vec3 eye_dir = normalize(start_camera - point);
	int t1, t2;
	glm::vec2 v1;
	glm::vec4 v2, v3;
	for (int i = 0; i < num_direct_light; ++i) {
		float intensity = glm::dot(-direct_lights[i], normal) * glm::dot(eye_dir, normal);
		if (intensity < 0)
			continue;
		float depth = tracing(point, -direct_lights[i], 100, t1, t2, v2, v1, v3, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
		if (depth < 1000)
			continue;
		color += intensity * (orig_color * direct_lights_color[i] * kd
			+ clamp((float)pow(glm::dot(glm::reflect(direct_lights[i], normal), eye_dir), 20), 0.0f, 1.f) * ks * direct_lights_color[i]);
	}
	for (int i = 0; i < num_point_light; ++i) {
		glm::vec3 dis = point - point_lights[i];
		float len = glm::length(dis);
		float l = 1 / (len * len);
		dis = normalize(dis);
		float intensity = glm::dot(-dis, normal) * glm::dot(eye_dir, normal);
		if (intensity < 0)
			continue;
		float depth = tracing(point, -dis, len, t1, t2, v2, v1, v3, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
		if (depth < len)
			continue;
		glm::vec3 para = kd * l * point_lights_color[i];
		color = color + intensity * (orig_color * para
			+ clamp((float)pow(dot(reflect(dis, normal), eye_dir), alpha), 0.f, 1.f) * ks * point_lights_color[i]);
	}
	return color;
}


__global__ void
render(unsigned int *g_odata, int imgw, int imgh,
glm::vec3 cam_up, glm::vec3 cam_forward, glm::vec3 right, glm::vec3 cam_pos, float dis_per_pix,
	InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object,
	int num_direct_lights, glm::vec3* direct_lights, glm::vec3* direct_lights_color,
	int num_point_lights, glm::vec3* point_lights, glm::vec3* point_lights_color, glm::vec3 ambient,
	uchar3* imagesBuffer, glm::ivec3* imageOffsetBuffer)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	glm::vec3 ray_p = cam_pos;
	glm::vec3 ray_d = glm::normalize(cam_forward + (x - imgw / 2) * dis_per_pix * right + (y - imgh / 2) * dis_per_pix * cam_up);
	glm::vec3 color(0, 0, 0);
	int tri_index, obj_index;

	int path_state[PATH_DEPTH];
	int mat_stack[PATH_DEPTH];
	glm::vec3 light_stack[PATH_DEPTH];
	glm::vec3 color_stack[PATH_DEPTH];
	glm::vec3 from_stack[PATH_DEPTH];
	glm::vec3 to_stack[PATH_DEPTH];
	glm::vec3 normal_stack[PATH_DEPTH];
	int node = 0;
	path_state[node] = 0;
	from_stack[node] = ray_p;
	to_stack[node] = ray_d;
	color_stack[node] = glm::vec3(0, 0, 0);
	light_stack[node] = glm::vec3(0, 0, 0);
	float nr;
	int hit_mat = 0;
	glm::vec4 hit_point;
	glm::vec2 uv;
	glm::vec4 normal;
	glm::vec3 normal3;
	glm::vec3 hit_point3;
	while (node >= 0) {
		if (path_state[node] == 0) {
			path_state[node] = 1;
			float depth;
			depth = tracing(from_stack[node], to_stack[node], -1, tri_index, obj_index, hit_point, uv, normal, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
			if (depth < 1e20) {
				hit_point3 = glm::vec3(hit_point.x, hit_point.y, hit_point.z);
				normal3 = glm::vec3(normal.x, normal.y, normal.z);
				glm::vec3 orig_color;
				light_stack[node] = lighting(from_stack[node], hit_point3, normal3, tri_index, uv, obj_index, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object,
					num_direct_lights, direct_lights, direct_lights_color, num_point_lights, point_lights, point_lights_color, ambient, imagesBuffer, imageOffsetBuffer, orig_color);
				color_stack[node] = orig_color;
				normal_stack[node] = normal3;
				ray_d = to_stack[node];
				to_stack[node] = hit_point3;
				mat_stack[node] = obj_index;
				float kr = instanceData[obj_index].kr;
				if (kr > 0 && node < PATH_DEPTH - 1) {
					node += 1;
					path_state[node] = 0;
					from_stack[node] = hit_point3;
					to_stack[node] = ray_d - 2 * glm::dot(ray_d, normal3) * normal3;
					light_stack[node] = glm::vec3(0, 0, 0);
					continue;
				}
			}
			else {
				path_state[node] = 3;
			}
		}
		if (path_state[node] == 1) {
			path_state[node] = 2;
			obj_index = mat_stack[node];
			float kf = instanceData[obj_index].kf;
			if (kf > 0 && node < PATH_DEPTH - 1) {
				nr = instanceData[obj_index].nr;
				normal3 = normal_stack[node];
				ray_d = glm::normalize(to_stack[node] - from_stack[node]);
				float cost = glm::dot(normal3, ray_d);
				if (cost < 0) {
					nr = 1 / nr;
					cost = -cost;
				}
				else {
					normal = -normal;
				}
				float rootContent = 1 - nr * nr * (1 - cost * cost);
				if (rootContent >= 0) {
					rootContent = sqrt(rootContent);
					node += 1;
					path_state[node] = 0;
					from_stack[node] = to_stack[node - 1];
					to_stack[node] = (nr * cost - rootContent) * normal3 + nr * ray_d;
					light_stack[node] = glm::vec3(0, 0, 0);
					continue;
				}
			}
		}
		if (path_state[node] == 2) {
			path_state[node] = 3;
			obj_index = mat_stack[node];
			/*float ks = texture2D(materialSampler, vec2(1.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
			if (hit_mat < use_path && node < PATH_DEPTH - 1 && ks > 0) {
				normal = normal_stack[node];
				ray_t = normalize(to_stack[node] - from_stack[node]);
				float alpha = texture2D(materialSampler, vec2(20.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
				hit_mat += 1;
				node += 1;
				path_state[node] = 0;
				from_stack[node] = to_stack[node - 1];
				to_stack[node] = phoneSample(ray_t, normal, alpha);
				continue;
			}*/
		}
		if (path_state[node] == 3) {
			if (node == 0)
				break;
			int obj_index = mat_stack[node - 1];
			if (path_state[node - 1] == 1) {
				light_stack[node - 1] = (1 - instanceData[obj_index].kr) * light_stack[node - 1] 
					+ instanceData[obj_index].kr * color_stack[node - 1] * light_stack[node] / 255.0f;
			}
			else
				if (path_state[node - 1] == 2) {
					light_stack[node - 1] = (1 - instanceData[obj_index].kf) * light_stack[node - 1]
						+ instanceData[obj_index].kf * color_stack[node - 1] * light_stack[node] / 255.0f;
				}
				else {
					hit_mat -= 1;
					normal3 = normal_stack[node - 1];
					ray_d = glm::normalize(to_stack[node - 1] - from_stack[node - 1]);
					float alpha = instanceData[obj_index].alpha;
					light_stack[node - 1] = (1 - instanceData[obj_index].ks) * light_stack[node - 1]
						+ instanceData[obj_index].ks * color_stack[node - 1] * light_stack[node] * glm::dot(-ray_d, normal3) / 255.0f;
				}
				node -= 1;
		}
	}

	uchar4 c4 = make_uchar4(light_stack[0].r, light_stack[0].g, light_stack[0].b, 255);
	g_odata[y*imgw + x] = rgbToInt(c4.x, c4.y, c4.z);
}

extern "C" void
cudaRender(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw, int imgh)
{
	static float count = 1;
	float dis_per_pix = tan(World::fov * 0.5 * 3.141592654 / 180.0) / (imgw / 2);
	glm::vec3 right = glm::cross(World::camera_lookat, World::camera_up);
	render << < grid, block, sbytes >> >(g_odata, imgw, imgh,
		World::camera_up, World::camera_lookat, right, World::camera, dis_per_pix,
		g_world.materialBuffer, g_world.vertexBuffer, g_world.normalBuffer, g_world.texBuffer, g_world.num_objects,
		g_world.lights.direct_light_dir.size(), g_world.directLightsBuffer, g_world.directLightsColorBuffer,
		g_world.lights.point_light_pos.size(), g_world.pointLightsBuffer, g_world.pointLightsColorBuffer, g_world.lights.ambient * count,
		g_world.texImagesBuffer, g_world.texOffsetBuffer);
	
}