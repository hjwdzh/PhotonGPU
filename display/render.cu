#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// Utilities and system includes

#include <hip/hip_runtime_api.h>
#include "../scene/world.h"

#define PATH_DEPTH 10
#define PATH_MAX_DEPTH 3
#define NUM_SAMPLE 1
#define CAUSTIC_X_MIN -12.8
#define CAUSTIC_MAP_DIS 0.05

__device__ __host__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}

__device__ glm::vec3 Inttorgb(int x)
{
	glm::vec3 rgb;
	rgb.b = (x >> 16);
	rgb.g = (x >> 8) & 0xff;
	rgb.r = (x & 0xff);
	return rgb;
}

__device__ __host__
float rayIntersectsTriangle(glm::vec3& p, glm::vec3& d,
glm::vec3& v0, glm::vec3& v1, glm::vec3& v2, float& u, float& v) {

	glm::vec3 e1 = v1 - v0;
	glm::vec3 e2 = v2 - v0;
	glm::vec3 h = glm::cross(d, e2);
	float a = glm::dot(e1, h);

	if (a > -0.00001 && a < 0.00001)
		return -1;

	float f = 1 / a;
	glm::vec3 s = p - v0;
	u = f * glm::dot(s, h);

	if (u < -1e-3 || u > 1 + 1e-3)
		return -1;

	glm::vec3 q = glm::cross(s, e1);
	v = f * glm::dot(d, q);

	if (v < -1e-3 || u + v > 1 + 1e-3)
		return -1;

	// at this stage we can compute t to find out where
	// the intersection point is on the line
	float t = f * glm::dot(e2, q);

	if (t > 0.00001) // ray intersection
		return t;

	else // this means that there is a line intersection
		// but not a ray intersection
		return -1;
}

__device__ __host__
void swaps(float& x, float& y) {
	float t = x;
	x = y, y = t;
}

__device__ __host__
int BoundingBoxIntersect(glm::vec3& ray_o, glm::vec3& ray_t, glm::vec3& minP, glm::vec3& maxP) {
	auto r = ray_t + glm::vec3(1e-6, 1e-6, 1e-6);
	auto rinv = glm::vec3(1 / r.x, 1 / r.y, 1 / r.z);
	float tx1 = (minP.x - ray_o.x)*rinv.x;
	float tx2 = (maxP.x - ray_o.x)*rinv.x;
	float tmin, tmax;
	if (rinv.x > 0)
		tmin = tx1, tmax = tx2;
	else
		tmin = tx2, tmax = tx1;

	float ty1 = (minP.y - ray_o.y)*rinv.y;
	float ty2 = (maxP.y - ray_o.y)*rinv.y;

	if (rinv.y > 0)
		tmin = max(tmin, ty1),
		tmax = min(tmax, ty2);
	else
		tmin = max(tmin, ty2),
		tmax = min(tmax, ty1);

	float tz1 = (minP.z - ray_o.z)*rinv.z;
	float tz2 = (maxP.z - ray_o.z)*rinv.z;

	if (rinv.z > 0)
		tmin = max(tmin, tz1),
		tmax = min(tmax, tz2);
	else
		tmin = max(tmin, tz2),
		tmax = min(tmax, tz1);
	return tmax >= tmin;

}

__device__ __host__
float tracing(glm::vec3& ray_o, glm::vec3& ray_t, float shadow, int& tri, int& obj, glm::vec3& hit_point, glm::vec2& uv, glm::vec3& normal,
InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object) {
	float depth = 1e30;
	obj = -1;
	tri = -1;
	int j = 0;
	for (int k = 0; k < num_object; ++k) {
		int next_object = instanceData[k].s;
		if ((k < 1 || k >= 7) && !BoundingBoxIntersect(ray_o, ray_t, instanceData[k].minPos, instanceData[k].maxPos)) {
			j = next_object;
			continue;
		}
		while (j < next_object) {
			glm::vec3& v1 = vertexBuffer[j];
			glm::vec3& v2 = vertexBuffer[j + 1];
			glm::vec3& v3 = vertexBuffer[j + 2];
			float u, v;
			float t = rayIntersectsTriangle(ray_o, ray_t, v1, v2, v3, u, v);
			if (t > 1e-2 && t < depth) {
				depth = t;
				hit_point = ((ray_o + ray_t * depth));
				if (shadow >= 0) {
					if (t < shadow) {
						return t;
					}
				}
				else {
					obj = k;
					tri = j;
					glm::vec3& n1 = normalBuffer[j];
					glm::vec3& n2 = normalBuffer[j + 1];
					glm::vec3& n3 = normalBuffer[j + 2];
					normal = u * (n2 - n1) + v * (n3 - n1) + n1;
					glm::vec2& uv1 = texBuffer[j];
					glm::vec2& uv2 = texBuffer[j + 1];
					glm::vec2& uv3 = texBuffer[j + 2];
					uv = uv1 + u * (uv2 - uv1) + v * (uv3 - uv1);
				}
			}
			j += 3;
		}
	}
	normal = normalize(normal);
	return depth;
}

__device__ __host__
glm::vec3 fetchTex(glm::vec2& uv, int objIndex, uchar3* imagesBuffer, glm::ivec3* imageOffsetBuffer)
{
	glm::ivec3& info = imageOffsetBuffer[objIndex];
	int offset = info.x;
	int iy = info.y;
	int ix = info.z;
	float x = ix * uv.x;
	float y = iy * uv.y;
	int lx = x, ly = y;
	int rx = lx + 1, ry = ly + 1;
	float wx = x - lx, wy = y - ly;
	if (lx < 0)
		lx += wx;
	if (ly < 0)
		ly += wy;
	if (rx >= ix)
		rx -= ix;
	if (ry >= iy)
		ry -= iy;
	int ind1 = offset + ly * ix + lx;
	int ind2 = offset + ly * ix + rx;
	int ind3 = offset + ry * ix + lx;
	int ind4 = offset + ry * ix + rx;
	uchar3& c1 = imagesBuffer[ind1];
	uchar3& c2 = imagesBuffer[ind2];
	uchar3& c3 = imagesBuffer[ind3];
	uchar3& c4 = imagesBuffer[ind4];
	float cx = (c1.x * (1 - wx) + c2.x * wx) * (1 - wy) + (c3.x * (1 - wx) + c4.x * wx) * wy;
	float cy = (c1.y * (1 - wx) + c2.y * wx) * (1 - wy) + (c3.y * (1 - wx) + c4.y * wx) * wy;
	float cz = (c1.z * (1 - wx) + c2.z * wx) * (1 - wy) + (c3.z * (1 - wx) + c4.z * wx) * wy;
	return glm::vec3(cz, cy, cx);
}

__device__ __host__
void projectCaustic(glm::vec3& ray_o, glm::vec3& ray_t, glm::vec3 &color,
InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object,
glm::vec3& light, glm::vec2& coords, uchar3* texImages, glm::ivec3* imageOffsets) {
	int tri_index, obj_index;
	glm::vec3 hit_point, normal;
	glm::vec2 uv;
	float depth = tracing(ray_o, ray_t, -1, tri_index, obj_index, hit_point, uv, normal, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
	glm::vec3 orig_color = fetchTex(uv, obj_index, texImages, imageOffsets) / 255.0f;
	int steps = 0;
	float intensity = 1;
	while (depth < 1e20 && (instanceData[obj_index].kr != 0 || instanceData[obj_index].kf != 0)) {
		if (instanceData[obj_index].kf != 0) {
			float nr = instanceData[obj_index].nr;
			float cost = glm::dot(normal, ray_t);
			if (cost < 0) {
				nr = 1 / nr;
				cost = -cost;
			}
			else {
				normal = -normal;
			}
			float rootContent = 1 - nr * nr * (1 - cost * cost);
			if (rootContent >= 0) {
				ray_o = glm::vec3(hit_point.x, hit_point.y, hit_point.z);
				ray_t = (nr * cost - sqrt(rootContent)) * normal + nr * ray_t;
				intensity *= instanceData[obj_index].kf;
			}
			else {
				ray_o = glm::vec3(hit_point.x, hit_point.y, hit_point.z);
				ray_t = glm::reflect(ray_t, glm::vec3(normal.x, normal.y, normal.z));
			}
		}
		else if (instanceData[obj_index].kr != 0) {
			ray_o = glm::vec3(hit_point.x, hit_point.y, hit_point.z);
			ray_t = glm::reflect(ray_t, glm::vec3(normal.x, normal.y, normal.z));
			intensity *= instanceData[obj_index].kr;
		}
		steps++;
		if (steps > 2)
			break;
		depth = tracing(ray_o, ray_t, -1, tri_index, obj_index, hit_point, uv, normal, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
	}
	if (obj_index == 0 && steps > 0) {
		float x = (hit_point.x - CAUSTIC_X_MIN) / CAUSTIC_MAP_DIS;
		float y = (hit_point.z - CAUSTIC_X_MIN) / CAUSTIC_MAP_DIS;
		coords = glm::vec2(x, y);
		light = intensity * color * orig_color;
	}
	else {
		coords = glm::vec2(-1, -1);
		light = glm::vec3(0, 0, 0);
	}
}


__device__ __host__
glm::vec3 lighting(glm::vec3& start_camera, glm::vec3& point, glm::vec3& normal, int tri_index, glm::vec2& uv, int obj_index,
InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object,
int num_direct_light, glm::vec3* direct_lights, glm::vec3* direct_lights_color,
int num_point_light, glm::vec3* point_lights, glm::vec3* point_lights_color, glm::vec3& ambient,
uchar3* imagesBuffer, glm::ivec3* imageOffsetBuffer, glm::vec3& orig_color, glm::vec3* causticMap) {
	float kd = instanceData[obj_index].kd;
	float ks = instanceData[obj_index].ks;//texture2D(materialSampler, vec2(1.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
	float ka = instanceData[obj_index].ka;// texture2D(materialSampler, vec2(16.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
	float alpha = instanceData[obj_index].alpha;// texture2D(materialSampler, vec2(20.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;

	orig_color = fetchTex(uv, obj_index, imagesBuffer, imageOffsetBuffer);
	//	int tex = int(0.1 + texture2D(materialSampler, vec2(2.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r);
	//	orig_color = texture2D(renderSampler[tex], uv).rgb;
	glm::vec3 color = ka * orig_color * ambient;
	glm::vec3 eye_dir = normalize(start_camera - point);
	int t1, t2;
	glm::vec2 v1;
	glm::vec3 v2, v3;
	for (int i = 0; i < num_direct_light; ++i) {
		float intensity = glm::dot(-direct_lights[i], normal) * glm::dot(eye_dir, normal);
		if (intensity < 0)
			continue;
		float depth = tracing(point, -direct_lights[i], 100, t1, t2, v2, v1, v3, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
		if (depth < 1000) {
			continue;
		}
		color += intensity * (orig_color * direct_lights_color[i] * kd
			+ clamp((float)pow(glm::dot(glm::reflect(direct_lights[i], normal), eye_dir), alpha), 0.0f, 1.f) * ks * direct_lights_color[i]);
	}
	for (int i = 0; i < num_point_light; ++i) {
		glm::vec3 dis = point - point_lights[i];
		float len = glm::length(dis);
		float l = 1 / (len * len);
		dis = normalize(dis);
		float intensity = glm::dot(-dis, normal) * glm::dot(eye_dir, normal);
		if (intensity < 0)
			continue;
		float depth = tracing(point, -dis, len, t1, t2, v2, v1, v3, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
		if (depth < len)
			continue;
		glm::vec3 para = kd * l * point_lights_color[i];
		color = color + intensity * (orig_color * para
			+ clamp((float)pow(dot(reflect(dis, normal), eye_dir), alpha), 0.f, 1.f) * ks * point_lights_color[i]);
	}
	if (obj_index == 0) {
		int rx = (point.x - CAUSTIC_X_MIN) / CAUSTIC_MAP_DIS;
		int ry = (point.z - CAUSTIC_X_MIN) / CAUSTIC_MAP_DIS;
		if (rx < 512 && ry < 512 && rx >= 0 && ry >= 0) {
			glm::vec3 caustic = causticMap[ry * 512 + rx];
			color = color + glm::dot(eye_dir, normal) * kd * caustic;
			color.x = clamp(color.x, 0.0f, 255.f);
			color.y = clamp(color.y, 0.0f, 255.f);
			color.z = clamp(color.z, 0.0f, 255.f);
		}
	}
	return color;
}


__global__ void
render(unsigned int *g_odata, int imgw, int imgh,
glm::vec3 cam_up, glm::vec3 cam_forward, glm::vec3 right, glm::vec3 cam_pos, float dis_per_pix,
InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object,
int num_direct_lights, glm::vec3* direct_lights, glm::vec3* direct_lights_color,
int num_point_lights, glm::vec3* point_lights, glm::vec3* point_lights_color, glm::vec3 ambient,
uchar3* imagesBuffer, glm::ivec3* imageOffsetBuffer,
glm::vec3* causticMap)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	glm::vec3 ray_p = cam_pos;
	glm::vec3 ray_d = glm::normalize(cam_forward + (x - imgw / 2) * dis_per_pix * right + (y - imgh / 2) * dis_per_pix * cam_up);
	glm::vec3 color(0, 0, 0);
	int tri_index, obj_index;
	int path_state[PATH_DEPTH];
	int mat_stack[PATH_DEPTH];
	glm::vec3 light_stack[PATH_DEPTH];
	glm::vec3 color_stack[PATH_DEPTH];
	glm::vec3 from_stack[PATH_DEPTH];
	glm::vec3 to_stack[PATH_DEPTH];
	glm::vec3 normal_stack[PATH_DEPTH];
	int node = 0;
	path_state[node] = 0;
	from_stack[node] = ray_p;
	to_stack[node] = ray_d;
	color_stack[node] = glm::vec3(0, 0, 0);
	light_stack[node] = glm::vec3(0, 0, 0);
	float nr;
	int hit_mat = 0;
	glm::vec3 hit_point;
	glm::vec2 uv;
	glm::vec3 normal;
	while (node >= 0) {
		if (path_state[node] == 0) {
			path_state[node] = 1;
			float depth;
			depth = tracing(from_stack[node], to_stack[node], -1, tri_index, obj_index, hit_point, uv, normal, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
			if (depth < 1e20) {
				glm::vec3 orig_color;
				light_stack[node] = lighting(from_stack[node], hit_point, normal, tri_index, uv, obj_index, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object,
					num_direct_lights, direct_lights, direct_lights_color, num_point_lights, point_lights, point_lights_color, ambient,
					imagesBuffer, imageOffsetBuffer, orig_color, causticMap);
				color_stack[node] = orig_color;
				normal_stack[node] = normal;
				ray_d = to_stack[node];
				to_stack[node] = hit_point;
				mat_stack[node] = obj_index;
				float kr = instanceData[obj_index].kr;
				if (kr > 0 && node < PATH_DEPTH - 1) {
					color_stack[node] = instanceData[obj_index].kr * color_stack[node];
					node += 1;
					path_state[node] = 0;
					from_stack[node] = hit_point;
					to_stack[node] = ray_d - 2 * glm::dot(ray_d, normal) * normal;
					light_stack[node] = glm::vec3(0, 0, 0);
					continue;
				}
			}
			else {
				path_state[node] = 3;
			}
		}
		if (path_state[node] == 1) {
			path_state[node] = 2;
			obj_index = mat_stack[node];
			float kf = instanceData[obj_index].kf;
			if (kf > 0 && node < PATH_DEPTH - 1) {
				nr = instanceData[obj_index].nr;
				normal = normal_stack[node];
				ray_d = glm::normalize(to_stack[node] - from_stack[node]);
				float cost = glm::dot(normal, ray_d);
				if (cost < 0) {
					nr = 1 / nr;
					cost = -cost;
				}
				else {
					normal = -normal;
				}
				float rootContent = 1 - nr * nr * (1 - cost * cost);
				if (rootContent >= 0) {
					color_stack[node] = instanceData[obj_index].kf * color_stack[node];
					rootContent = sqrt(rootContent);
					node += 1;
					path_state[node] = 0;
					from_stack[node] = to_stack[node - 1];
					to_stack[node] = (nr * cost - rootContent) * normal + nr * ray_d;
					light_stack[node] = glm::vec3(0, 0, 0);
					continue;
				}
				else {
					float kr = 1;
					if (kr > 0 && node < PATH_DEPTH - 1) {
						light_stack[node] = glm::vec3(0, 0, 0);
						node += 1;
						path_state[node] = 0;
						from_stack[node] = to_stack[node - 1];
						to_stack[node] = ray_d - 2 * glm::dot(ray_d, normal) * normal;
						light_stack[node] = glm::vec3(0, 0, 0);
						continue;
					}
					else {
						g_odata[y*imgw + x] = 0;
						return;
					}
				}
			}
		}
		if (path_state[node] == 2) {
			path_state[node] = 3;
			obj_index = mat_stack[node];
			/*float ks = texture2D(materialSampler, vec2(1.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
			if (hit_mat < use_path && node < PATH_DEPTH - 1 && ks > 0) {
			normal = normal_stack[node];
			ray_t = normalize(to_stack[node] - from_stack[node]);
			float alpha = texture2D(materialSampler, vec2(20.5 / MATERIAL_LEN, (obj_index + 0.5) / num_object)).r;
			hit_mat += 1;
			node += 1;
			path_state[node] = 0;
			from_stack[node] = to_stack[node - 1];
			to_stack[node] = phoneSample(ray_t, normal, alpha);
			continue;
			}*/
		}
		if (path_state[node] == 3) {
			if (node == 0)
				break;
			int obj_index = mat_stack[node - 1];
			if (path_state[node - 1] == 1) {
				light_stack[node - 1] = (1 - instanceData[obj_index].kr) * light_stack[node - 1]
					+ color_stack[node - 1] * light_stack[node] / 255.0f;
			}
			else
				if (path_state[node - 1] == 2) {
					light_stack[node - 1] = (1 - instanceData[obj_index].kf) * light_stack[node - 1]
						+ color_stack[node - 1] * light_stack[node] / 255.0f;
				}
				else {
					hit_mat -= 1;
					normal = normal_stack[node - 1];
					ray_d = glm::normalize(to_stack[node - 1] - from_stack[node - 1]);
					float alpha = instanceData[obj_index].alpha;
					light_stack[node - 1] = (1 - instanceData[obj_index].ks) * light_stack[node - 1]
						+ instanceData[obj_index].ks * color_stack[node - 1] * light_stack[node] * glm::dot(-ray_d, normal) / 255.0f;
				}
				node -= 1;
		}
	}

	uchar4 c4 = make_uchar4(light_stack[0].r, light_stack[0].g, light_stack[0].b, 255);
	g_odata[y*imgw + x] = rgbToInt(c4.x, c4.y, c4.z);
}

__global__ void
filter(unsigned int *g_odata, int imgw, int imgh) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;
	int id = y * imgw + x;
	if (g_odata[id] == 0) {
		glm::vec3 rgb(0, 0, 0);
		int count = 0;
		for (int dx = -5; dx <= 5; ++dx) {
			for (int dy = -5; dy <= 5; ++dy) {
				int nx = x + dx;
				int ny = y + dy;
				if (nx >= 0 && nx < imgw && ny >= 0 && ny < imgh) {
					int nid = ny * imgw + nx;
					if (g_odata[nid] != 0) {
						count += 1;
						rgb += Inttorgb(g_odata[nid]);
					}
				}
			}
		}
		if (count > 0)
			g_odata[id] = rgbToInt(rgb.r / count, rgb.g / count, rgb.b / count);
		else
			g_odata[id] = rgbToInt(255, 0, 0);
	}
}


__global__ void
ClearCausticMap(glm::ivec3 *g_odata, int imgw, int imgh) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;
	g_odata[y * imgw + x] = glm::ivec3(0, 0, 0);
}

__global__ void
CausticRender(glm::vec3 *causticMap, glm::vec2* cuasticCoords, int imgw, int imgh,
InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object,
glm::vec3 dir, glm::vec3 color, uchar3* texImages, glm::ivec3* imageOffsets) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;
	glm::vec3 point(x * CAUSTIC_MAP_DIS + CAUSTIC_X_MIN, 0, y * CAUSTIC_MAP_DIS + CAUSTIC_X_MIN);
	projectCaustic(point - dir * 1000.0f, dir, color, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object,
		causticMap[y * imgw + x], cuasticCoords[y * imgw + x], texImages, imageOffsets);
}


__global__ void
combineCaustic(unsigned int *g_odata, glm::ivec3* causticMap, int imgw, int imgh,
glm::vec3 cam_up, glm::vec3 cam_forward, glm::vec3 right, glm::vec3 cam_pos, float dis_per_pix,
InstanceData* instanceData, glm::vec3* vertexBuffer, glm::vec3* normalBuffer, glm::vec2* texBuffer, int num_object) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	glm::vec3 ray_p = cam_pos;
	glm::vec3 ray_d = glm::normalize(cam_forward + (x - imgw / 2) * dis_per_pix * right + (y - imgh / 2) * dis_per_pix * cam_up);
	glm::vec3 color(0, 0, 0);
	int tri_index, obj_index;

	glm::vec3 hit_point, normal;
	glm::vec2 uv;
	float depth;
	depth = tracing(ray_p, ray_d, -1, tri_index, obj_index, hit_point, uv, normal, instanceData, vertexBuffer, normalBuffer, texBuffer, num_object);
	if (obj_index == 0) {
		int rx = (hit_point.x - CAUSTIC_X_MIN) / CAUSTIC_MAP_DIS;
		int ry = (hit_point.z - CAUSTIC_X_MIN) / CAUSTIC_MAP_DIS;
		if (rx < 512 && ry < 512 && rx >= 0 && ry >= 0) {
			auto& p = causticMap[ry * 512 + rx];
			glm::vec3 np = Inttorgb(g_odata[y * imgw + x]);
			np += p;
			np.x = clamp(np.x, 0.f, 255.f);
			np.y = clamp(np.y, 0.f, 255.f);
			np.z = clamp(np.z, 0.f, 255.f);
			np = glm::vec3(255, 0, 0);
			g_odata[y * imgw + x] = rgbToInt(p.x, p.y, p.z);
		}
	}
}

__global__ void
SplatCaustic(glm::vec3* caustics, glm::vec2* causticCoords, glm::ivec3* causticMaps, int imgw, int imgh) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;
	int ry = y + 1, rx = x + 1;
	if (ry < imgh && rx < imgw) {
		int id[4];
		id[0] = y * imgw + x;
		id[1] = id[0] + 1;
		id[2] = id[0] + imgw;
		id[3] = id[2] + 1;
		float minX = 1e20f, maxX = -1e20f, minY = 1e20f, maxY = -1e20f;
		for (int i = 0; i < 4; ++i) {
			auto& p = causticCoords[id[i]];
			if (causticCoords[id[i]].x < 0)
				return;
			if (p.x < minX)
				minX = p.x;
			if (p.x > maxX)
				maxX = p.x;
			if (p.y < minY)
				minY = p.y;
			if (p.y > maxY)
				maxY = p.y;
		}
		if (maxX - minX > 15 || maxY - minY > 15)
			return;
		int stepX = (maxX - minX) + 1;
		int stepY = (maxY - minY) + 1;
		int steps;
		if (stepX > stepY)
			steps = stepX;
		else
			steps = stepY;
		if (steps == 1)
			steps += 1;
		//		steps *= 2;
		float weight = 255.0 / (steps * steps);
		float stepW = 1.0 / (steps - 1);
		for (int i = 0; i < steps; ++i) {
			for (int j = 0; j < steps; ++j) {
				float wx = stepW * j;
				float wy = stepW * i;
				glm::vec3 interp = (caustics[id[0]] * (1 - wx) + caustics[id[1]] * wx) * (1 - wy)
					+ (caustics[id[2]] * (1 - wx) + caustics[id[3]] * wx) * wy;
				glm::vec2 interp_coords = (causticCoords[id[0]] * (1 - wx) + causticCoords[id[1]] * wx) * (1 - wy)
					+ (causticCoords[id[2]] * (1 - wx) + causticCoords[id[3]] * wx) * wy;
				int nx = interp_coords.x, ny = interp_coords.y;
				if (nx >= 0 && nx < imgw && ny >= 0 && ny < imgh) {
					atomicAdd(&causticMaps[ny * imgw + nx].x, interp.x * weight);
					atomicAdd(&causticMaps[ny * imgw + nx].y, interp.y * weight);
					atomicAdd(&causticMaps[ny * imgw + nx].z, interp.z * weight);
				}
			}
		}
	}
}

__global__ void
FilterCaustic(glm::ivec3* causticMap, glm::vec3* causticBuffer, int imgw, int imgh) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;
	int id = y * imgw + x;
	auto& pix = causticMap[id];
	int temp[3][3] =
	{
		{ 1, 2, 1 },
		{ 2, 4, 2 },
		{ 1, 2, 1 }
	};
	if (pix.x == 0 && pix.y == 0 && pix.z == 0 || true) {
		glm::ivec4 pt;
		for (int py = y - 1; py <= y + 1; ++py) {
			if (py < 0 || py >= imgh)
				continue;
			for (int px = x - 1; px <= x + 1; ++px) {
				if (px < 0 || px >= imgw)
					continue;
				int dy = py - y + 1;
				int dx = px - x + 1;
				auto& p = causticMap[py * imgw + px];
				if (p.x != 0 || p.y != 0 || p.z != 0) {
					pt += glm::ivec4(p, 1) * temp[dy][dx];
				}
			}
		}
		if (pt.w > 0)
			causticBuffer[id] = glm::vec3((float)pt.x / pt.w, (float)pt.y / pt.w, (float)pt.z / pt.w);
		else
			causticBuffer[id] = glm::vec3(0, 0, 0);
	}
	else {
		causticBuffer[id] = glm::vec3(pix.x, pix.y, pix.z);
	}
}

extern "C" void
cudaRender(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw, int imgh)
{
	static float count = 1;
	float dis_per_pix = tan(World::fov * 0.5 * 3.141592654 / 180.0) / (imgw / 2);
	glm::vec3 right = glm::cross(World::camera_lookat, World::camera_up);
	ClearCausticMap << < grid, block, sbytes >> >(g_world.causticMapBuffer, imgw, imgh);
	for (int i = 0; i < g_world.lights.direct_light_dir.size(); ++i) {
		CausticRender << < grid, block, sbytes >> > (g_world.causticBuffer, g_world.causticCoordsBuffer, imgw, imgh,
			g_world.materialBuffer, g_world.vertexBuffer, g_world.normalBuffer, g_world.texBuffer, g_world.num_objects,
			g_world.lights.direct_light_dir[i], g_world.lights.direct_light_color[i], g_world.texImagesBuffer, g_world.texOffsetBuffer);
		SplatCaustic << < grid, block, sbytes >> > (g_world.causticBuffer, g_world.causticCoordsBuffer, g_world.causticMapBuffer, imgw, imgh);
		FilterCaustic << < grid, block, sbytes >> > (g_world.causticMapBuffer, g_world.causticBuffer, imgw, imgh);
	}

	render << < grid, block, sbytes >> >(g_odata, imgw, imgh,
		World::camera_up, World::camera_lookat, right, World::camera, dis_per_pix,
		g_world.materialBuffer, g_world.vertexBuffer, g_world.normalBuffer, g_world.texBuffer, g_world.num_objects,
		g_world.lights.direct_light_dir.size(), g_world.directLightsBuffer, g_world.directLightsColorBuffer,
		g_world.lights.point_light_pos.size(), g_world.pointLightsBuffer, g_world.pointLightsColorBuffer, g_world.lights.ambient * count,
		g_world.texImagesBuffer, g_world.texOffsetBuffer,
		g_world.causticBuffer);
	//	filter << < grid, block, sbytes >> >(g_odata, imgw, imgh);
	/*	combineCaustic << < grid, block, sbytes >> >(g_odata, g_world.causticMapBuffer, imgw, imgh,
	World::camera_up, World::camera_lookat, right, World::camera, dis_per_pix,
	g_world.materialBuffer, g_world.vertexBuffer, g_world.normalBuffer, g_world.texBuffer, g_world.num_objects);*/
}